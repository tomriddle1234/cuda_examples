#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

//  Windows
#ifdef _WIN32
#include <Windows.h>
double get_wall_time(){
	LARGE_INTEGER time, freq;
	if (!QueryPerformanceFrequency(&freq)){
		//  Handle error
		return 0;
	}
	if (!QueryPerformanceCounter(&time)){
		//  Handle error
		return 0;
	}
	return (double)time.QuadPart / freq.QuadPart;
}
double get_cpu_time(){
	FILETIME a, b, c, d;
	if (GetProcessTimes(GetCurrentProcess(), &a, &b, &c, &d) != 0){
		//  Returns total user time.
		//  Can be tweaked to include kernel times as well.
		return
			(double)(d.dwLowDateTime |
			((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
	}
	else{
		//  Handle error
		return 0;
	}
}

//  Posix/Linux
#else
#include <sys/time.h>
double get_wall_time(){
	struct timeval time;
	if (gettimeofday(&time, NULL)){
		//  Handle error
		return 0;
	}
	return (double)time.tv_sec + (double)time.tv_usec * .000001;
}
double get_cpu_time(){
	return (double)clock() / CLOCKS_PER_SEC;
}
#endif

static void HandleError(hipError_t err,
	const char *file,
	int line)
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


const int arraySize = 1000000;
struct LargeData
{
	int hugevalue[arraySize] ;
};

int cc[arraySize];
int aa[arraySize];
int bb[arraySize];

void cpuAdd()
{
	//start cpu computation
	printf("Start CPU \n");

	for (size_t i = 0; i < arraySize; ++i)
	{
		aa[i] = i;
		bb[i] = i + 1;
	}

	double wall_time0, wall_time1;
	double cpu_time0, cpu_time1;

	wall_time0 = get_wall_time();
	cpu_time0 = get_cpu_time();

	for (size_t i = 0; i < arraySize; ++i)
		cc[i] = aa[i] + bb[i];

	wall_time1 = get_wall_time();
	cpu_time1 = get_cpu_time();

	printf("=== CPU ===\n");
	printf("CPU -- Wall time: %3.10f ms \n", (wall_time1 - wall_time0) * 1000);
	printf("CPU -- Cpu time: %3.10f ms \n", (cpu_time1 - cpu_time0) * 1000);
}

__global__
void addall(LargeData * a, LargeData * b, LargeData * c)
{
	int i = threadIdx.x;
	int j = blockIdx.x ;
	c->hugevalue[j * blockDim.x + i] = a->hugevalue[j * blockDim.x + i] + b->hugevalue[j * blockDim.x + i];
}


void launchAdd(LargeData * a, LargeData * b, LargeData * c)
{
	float time;
	double wall_time0, wall_time1;
	double cpu_time0, cpu_time1;
	hipEvent_t start, stop;

	dim3 threadsPerBlock(256, 1);
	dim3 numBlocks(arraySize / threadsPerBlock.x + 1, 1);

	wall_time0 = get_wall_time();
	cpu_time0 = get_cpu_time();

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	addall << <numBlocks, threadsPerBlock >> >(a, b, c);
	hipDeviceSynchronize();

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

	wall_time1 = get_wall_time();
	cpu_time1 = get_cpu_time();

	printf("=== CUDA Execution Time: ===\n");
	printf("Cuda event time to generate:  %3.10f ms \n", time);
	printf("Wall time: %3.10f ms \n", (wall_time1 - wall_time0) * 1000);
	printf("Cpu time: %3.10f ms \n", (cpu_time1 - cpu_time0) * 1000);
}

int main(void)
{

	cpuAdd();
	LargeData *a;
	LargeData *b;
	LargeData *c;

	hipMallocManaged((void**)&a, sizeof(LargeData));
	hipMallocManaged((void**)&b, sizeof(LargeData));
	hipMallocManaged((void**)&c, sizeof(LargeData));

	for (size_t i = 0; i < arraySize; ++i)
	{
		a->hugevalue[i] = i;
		b->hugevalue[i] = i+1;
		c->hugevalue[i] = 0;
	}
	
	launchAdd(a, b, c);

	for (size_t i = 0; i < 10; ++i)
		printf("%d,", c->hugevalue[i]);
	printf("\n");
	printf("%d\n", a->hugevalue[arraySize - 2]);
	printf("%d\n", b->hugevalue[arraySize - 2]);
	printf("%d\n" ,c->hugevalue[arraySize-2]);
	printf("%d\n", c->hugevalue[255]);
	printf("%d\n", c->hugevalue[256]);
	printf("CUDA Finished.\n");

	hipFree(a);
	hipFree(b);
	hipFree(c);

	hipDeviceReset();

	return 0;
}
